
#include <hip/hip_runtime.h>
__global__ void vecAdd_kernel(const int *A, const int *B, int *C)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	C[idx] = A[idx] + B[idx];
}

void vecAdd(const int *A, const int *B, int *C, int N)
{
	vecAdd_kernel <<< 1, N >>> (A, B, C);
}